#include "hip/hip_runtime.h"
#include <stdio.h>
#include "matrix.h"

void printMatrix(const Datatype* matrix, int size) {
    int size2 = size * size;
    for(int i = 0; i < size2; ++i){
        printf("%12.2lf ", matrix[i]);
        if(i % size == size - 1) printf("\n");
    }
    printf("\n");
}

 __global__ void multiplyMatrix(Datatype* matrix1, Datatype* matrix2, Datatype* matrix3, int size, int quantity, int offset){
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if(thread_index >= size*quantity){
        return;
    }
    // index = y * size + x;
    int x = thread_index % size;
    int y = thread_index / size;
    double sum = 0;
    for(int k = 0; k < size; ++k){
        sum += matrix1[y*size + k] * matrix2[k*size + x];
    }
    matrix3[thread_index] = sum;
 }

extern "C" void multiply(Datatype* matrix1, Datatype* matrix2, Datatype* matrix3, int size, int quantity, int offset) {
    Datatype* working1;
    Datatype* working2;
    Datatype* working3;
    hipMalloc((void **)&working1, size*quantity*sizeof(Datatype));
    hipMalloc((void **)&working2, size*size*sizeof(Datatype));
    hipMalloc((void **)&working3, size*quantity*sizeof(Datatype));


    hipMemcpy(working1, matrix1, size*quantity*sizeof(Datatype), hipMemcpyHostToDevice);
    hipMemcpy(working2, matrix2, size*size*sizeof(Datatype), hipMemcpyHostToDevice);

    int block_count = (size*quantity) / 32 + 1;
    multiplyMatrix<<<block_count,32>>>(working1, working2, working3, size, quantity, offset);
    // multiplyMatrix<<<block_count,32>>>(matrix1, matrix2, matrix3, size, quantity, offset);

    hipDeviceSynchronize();
    hipMemcpy(matrix3, working3, size*quantity*sizeof(Datatype), hipMemcpyDeviceToHost);

    hipFree(working1);
    hipFree(working2);
    hipFree(working3);
}
